#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 0;
    int t;
    scanf("%d",&t);
    while(t--)
    {
        scanf("%d",&numElements);

        size_t size = numElements * sizeof(float);
        //printf("[Vector addition of %d elements]\n", numElements);

        // Allocate the host input vector A
        float *h_A = (float *)malloc(size);

        // Allocate the host input vector B
        float *h_B = (float *)malloc(size);

        // Allocate the host output vector C
        float *h_C = (float *)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Initialize the host input vectors
        for (int i = 0; i < numElements; ++i)
        {
            scanf("%f",&h_A[i]);
         
        }
        for (int i = 0; i < numElements; ++i)
        {
         
            scanf("%f",&h_B[i]);
        }

        
        // Allocate the device input vector A
        float *d_A = NULL;
        err = hipMalloc((void **)&d_A, size);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Allocate the device input vector B
        float *d_B = NULL;
        err = hipMalloc((void **)&d_B, size);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Allocate the device output vector C
        float *d_C = NULL;
        err = hipMalloc((void **)&d_C, size);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
       // printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
        //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
       // printf("Copy output data from the CUDA device to the host memory\n");
        err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Verify that the result vector is correct
        for (int i = 0; i < numElements; ++i)
        {
            if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                exit(EXIT_FAILURE);
            }
        }

       // printf("Test PASSED\n");

        // Free device global memory
        err = hipFree(d_A);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipFree(d_B);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipFree(d_C);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        
        // Reset the device and exit
        // cudaDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling cudaDeviceReset causes all profile data to be
        // flushed before the application exits
        err = hipDeviceReset();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        for (int i = 0; i < numElements; ++i)
        {
            printf("%.2f ",h_C[i]);
        }       
        printf("\n");
	// Free host memory
        free(h_A);
        free(h_B);
        free(h_C);

        //printf("Done\n");
    
    }
    return 0;
}
